#include<stdio.h>
#include<hip/hip_runtime.h>
#include<time.h>
#include <cstring>
#include <ctime>
#include <stdlib.h>
#include <sys/stat.h>

__global__ void calc_freq(int *f , char *data)
{
	int id=blockIdx.x * blockDim.x + threadIdx.x;
	int x=0;
	if(data[id] >= 'a' && data[id] <= 'z')
	{
		x = data[id] - 'a';
		f[x]++;
	}	
}

__global__ void compress(char *h_r, char *a, char *en,char *h_lt)
{
	int id=blockIdx.x * blockDim.x + threadIdx.x;
	if(h_lt[id] == a[id])
	{
		for(int j=0;j<5;j++)
		{
			h_r += en[id+j];
		}
	}
}

int main()
{
	int *freq;
	int *h_freq;
	char *letter;
	char *h_letter;
	
	freq = (int*)malloc(26*sizeof(int));
	hipMalloc((void**)&h_freq, 26*sizeof(int));
	
	for(int i=0;i<26;i++)
	{
		freq[i] = 0;
	}
	
	FILE *fptr = fopen("input.txt" , "r");
	
	struct stat st; 
	int size;
     
    	if(stat("input.txt",&st)==0)
        	size = st.st_size;
    	else
        	size = -1;
        
        letter = (char*)malloc(size*sizeof(char));
	hipMalloc((void**)&h_letter, size*sizeof(char));
	int k=0;
	while(k<size)
	{
		letter[k] = fgetc(fptr);
		k++;
        	//printf("%c", *letter);
	}
	
	for(int i=0;i<size;i++)
		printf("%c",letter[i]);

	int threadsPerBlock;
	if (size<=1024) 
		threadsPerBlock=size;
	else 
		threadsPerBlock=1024;
		
    	int blocksPerGrid =(size + threadsPerBlock - 1) / threadsPerBlock;
	printf("\nblocksPerGrid=%d\n",blocksPerGrid);
	
	hipMemcpy(h_freq,freq,26*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(h_letter,letter,size*sizeof(char),hipMemcpyHostToDevice);
	
	calc_freq<<<threadsPerBlock,blocksPerGrid>>>(h_freq,h_letter);
	
	hipMemcpy(freq,h_freq,26*sizeof(int),hipMemcpyDeviceToHost);
	
	for(int i=0 ; i<26 ; i++)
	{
		printf("%d\n",freq[i]);
	}
	
	char arr[26];
	
	char c;
	int i;
	
	for( i=0, c = 'A'; i<26, c <= 'Z'; i++, c++)
	{
		arr[i] = c;
		printf("%c",arr[i]);
	}
	
	for(i=0;i<26;i++)
	{
		for(int j=i;j<26;j++)
		{
			int temp;
			char ctemp;
			
			if(freq[i] < freq[j])
			{
				temp = freq[i];
				freq[i] = freq[j];
				freq[j] = temp;
				
				ctemp = arr[i];
				arr[i] = arr[j];
				arr[j] = ctemp;
			}
		}
	}
	
	char encode[26][5] = {{'0'},{'0','0'},{'0','1'},{'0','0','0'},{'0','0','1'},{'0','1','0'},{'0','1','1'},{'0','0','0','0'},{'0','0','0','1'},{'0','0','1','0'},{'0','0','1','1'},{'0','1','0','0'},{'0','1','0','1'},{'0','1','1','0'},{'0','1','1','1'},{'0','0','0','0','0'},{'0','0','0','0','1'},{'0','0','0','1','0'},{'0','0','0','1','1'},{'0','0','1','0','0'},{'0','0','1','0','1'},{'0','0','1','1','0'},{'0','0','1','1','1'},{'0','1','0','0','0'},{'0','1','0','0','1'},{'0','1','0','1','0'}};
	
	for(int i=0 ; i<26 ; i++)
	{
		printf("%c %d ",arr[i],freq[i]);
		for(int j=0;j<5;j++)
		{
			printf("%c",encode[i][j]);
		}
		printf("\n");
	}
	
	fptr = fopen("output.txt","w");
	//FILE *h_fptr;
	char *h_arr;
	hipMalloc((void**)&h_arr, 26*sizeof(char));
	hipMemcpy(h_arr,arr,26*sizeof(char),hipMemcpyHostToDevice);
	char *h_en;
	hipMalloc((void**)&h_en, 26*5*sizeof(char));
	hipMemcpy(h_en,encode,26*5*sizeof(char),hipMemcpyHostToDevice);
	//cudaMemcpy(h_fptr,fptr,26*5*sizeof(char),cudaMemcpyHostToDevice);
	char *h_res;
	hipMalloc((void**)&h_res, 26*5*sizeof(char));
	compress<<<threadsPerBlock,blocksPerGrid>>>(h_res,h_arr,h_en,h_letter);
	char *res;
	hipMemcpy(res,h_res,26*5*sizeof(char),hipMemcpyDeviceToHost);
	for(int i=0 ; i<size*5;i++)
	{
		//printf("%c",res[i]);
		fprintf (fptr,"%c",res[i]);
	}
  	fclose(fptr);
	return 0;
}
